#include "hip/hip_runtime.h"
// CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>
// Utilities and system includes
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <stdio.h>
#include <iostream>
#include <random>
#include <math.h>

typedef unsigned int u32;
using namespace std;

#ifndef MIN
#define MIN(x,y) ((x < y) ? x : y)
#endif

unsigned int nextPow2(unsigned int x)
{
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}

void getNumBlocksAndThreads(int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
	threads = (n < maxThreads * 2) ? nextPow2((n + 1) / 2) : maxThreads;
	blocks = (n + (threads * 2 - 1)) / (threads * 2);
	blocks = MIN(maxBlocks, blocks);
}


// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
	__device__ inline operator T *()
	{
		extern __shared__ int __smem[];
		return (T *)__smem;
	}

	__device__ inline operator const T *() const
	{
		extern __shared__ int __smem[];
		return (T *)__smem;
	}
};

template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce6(T *g_idata, T *g_odata, unsigned int n)
{
	T *sdata = SharedMemory<T>();

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize * 2 + threadIdx.x;
	unsigned int gridSize = blockSize * 2 * gridDim.x;

	T mySum = 0;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n)
	{
		mySum += g_idata[i];

		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2 || i + blockSize < n)
			mySum += g_idata[i + blockSize];

		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = mySum;
	__syncthreads();


	// do reduction in shared mem
	if ((blockSize >= 512) && (tid < 256))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 256];
	}

	__syncthreads();

	if ((blockSize >= 256) && (tid < 128))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 128];
	}

	__syncthreads();

	if ((blockSize >= 128) && (tid < 64))
	{
		sdata[tid] = mySum = mySum + sdata[tid + 64];
	}

	__syncthreads();

	if (tid < 32)
	{
		// Fetch final intermediate sum from 2nd warp
		if (blockSize >= 64) mySum += sdata[tid + 32];
		// Reduce final warp using shuffle
		for (int offset = warpSize / 2; offset > 0; offset /= 2)
		{
			mySum += __shfl_down(mySum, offset);
		}
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = mySum;
}

__host__ __device__ __inline__ bool isPow2(unsigned int x)
{
	return ((x&(x - 1)) == 0);
}

__global__ void print(u32* in, const u32 n)
{
	const int gtid = blockDim.x*blockIdx.x + threadIdx.x;

	if (gtid == 1)
		printf("%d = %d\n", gtid, in[gtid]);
}

////////////////////////////////////////////////////////////////////////////////
//! Compute sum reduction on CPU
//! We use Kahan summation for an accurate sum of large arrays.
//! http://en.wikipedia.org/wiki/Kahan_summation_algorithm
//!
//! @param data       pointer to input data
//! @param size       number of input data elements
////////////////////////////////////////////////////////////////////////////////
template<class T>
T reduceCPU(T *data, int size)
{
	T sum = data[0];
	T c = (T)0.0;

	for (int i = 1; i < size; i++)
	{
		T y = data[i] - c;
		T t = sum + y;
		c = (t - sum) - y;
		sum = t;
	}

	return sum;
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void
reduce(int size, int threads, int blocks, T *d_idata, T *d_odata)
{
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	// when there is only one warp per block, we need to allocate two warps
	// worth of shared memory so that we don't index shared memory out of bounds
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

	//�����2������
	if (isPow2(size))
	{
		switch (threads)
		{
		case 512:
			reduce6<T, 512, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 256:
			reduce6<T, 256, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 128:
			reduce6<T, 128, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 64:
			reduce6<T, 64, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 32:
			reduce6<T, 32, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 16:
			reduce6<T, 16, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  8:
			reduce6<T, 8, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  4:
			reduce6<T, 4, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  2:
			reduce6<T, 2, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  1:
			reduce6<T, 1, true> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;
		}
	}
	else
	{
		switch (threads)
		{
		case 512:
			reduce6<T, 512, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 256:
			reduce6<T, 256, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 128:
			reduce6<T, 128, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 64:
			reduce6<T, 64, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 32:
			reduce6<T, 32, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case 16:
			reduce6<T, 16, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  8:
			reduce6<T, 8, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  4:
			reduce6<T, 4, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  2:
			reduce6<T, 2, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;

		case  1:
			reduce6<T, 1, false> << < dimGrid, dimBlock, smemSize >> >(d_idata, d_odata, size);
			break;
		}
	}
}

int main()
{
	//	�Ƿ�GPU����д��
	bool needReadBack = true;
	bool cpuFinalReduction = false;

	typedef u32 T;
	// ��Լ���鳤��
	const int size = 512;
	// ���blocks��
	const int maxBlocks = 64;
	// ÿblock������߳���
	const int maxThreads = 256;
	// ��������byte����
	u32 bytes = size * sizeof(u32);

	u32 sum = 0;
	int numBlocks = 0;
	int numThreads = 0;
	getNumBlocksAndThreads(size, maxBlocks, maxThreads, numBlocks, numThreads);

	cout << "size = " << size << endl;
	cout << "numBlocks = " << numBlocks << endl;
	cout << "numThreads = " << numThreads << endl;

	// ����host���飬������ֵ
	T *h_idata = (T *)malloc(bytes);
	// allocate mem for the result on host side
	T *h_odata = (T *)malloc(numBlocks*sizeof(T));

	//���������
	default_random_engine generator;
	uniform_int_distribution<u32> dis(0, 1000);
	for (u32 i = 0; i < size; ++i)
		h_idata[i] = dis(generator);

	// allocate device memory and data
	T *d_idata = NULL;
	T *d_odata = NULL;

	//	���豸�ڴ�������ռ�
	checkCudaErrors(hipMalloc((void **)&d_idata, bytes));
	checkCudaErrors(hipMalloc((void **)&d_odata, numBlocks*sizeof(T)));

	//	�������豸�ڴ�
	checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_odata, h_idata, numBlocks*sizeof(T), hipMemcpyHostToDevice));

	//	sampleʾ���� ----benchmarkReduce()----- ����
	T gpu_result = 0;

	//	������ʱ��
	StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);
	//	������ʱ��
	hipDeviceSynchronize();
	sdkStartTimer(&timer);

	// һ�׶ι�Լ�ں�
	reduce<T>(size, numThreads, numBlocks, d_idata, d_odata);

	// Clear d_idata for later use as temporary buffer.
	hipMemset(d_idata, 0, size*sizeof(T));

	// ��CPU�����Ĺ�Լ
	if (cpuFinalReduction)
	{
		// sum partial sums from each block on CPU
		// copy result from device to host
		checkCudaErrors(hipMemcpy(h_odata, d_odata, numBlocks*sizeof(T), hipMemcpyDeviceToHost));

		for (int i = 0; i < numBlocks; i++)
			gpu_result += h_odata[i];

		needReadBack = false;
	}
	else
	{
		// sum partial block sums on GPU
		int s = numBlocks;
		//	��block����������ʱ����CPU��Լ
		int cpuFinalThreshold = 1;

		while (s > cpuFinalThreshold)
		{
			int threads = 0, blocks = 0;
			//	���¼���thread����block��
			getNumBlocksAndThreads(s, maxBlocks, maxThreads, blocks, threads);
			hipMemcpy(d_idata, d_odata, s*sizeof(T), hipMemcpyDeviceToDevice);
			reduce<T>(s, threads, blocks, d_idata, d_odata);

			s = (s + (threads * 2 - 1)) / (threads * 2);
		}

		if (s > 1)
		{
			// copy result from device to host
			checkCudaErrors(hipMemcpy(h_odata, d_odata, s * sizeof(T), hipMemcpyDeviceToHost));

			for (int i = 0; i < s; i++)
			{
				gpu_result += h_odata[i];
			}

			needReadBack = false;
		}
	}

	hipDeviceSynchronize();
	sdkStopTimer(&timer);

	if (needReadBack)
	{
		// copy final sum from device to host
		checkCudaErrors(hipMemcpy(&gpu_result, d_odata, sizeof(T), hipMemcpyDeviceToHost));
	}

	double reduceTime = sdkGetAverageTimerValue(&timer) * 1e-3;

	// compute reference solution
	T cpu_result = reduceCPU<T>(h_idata, size);

	printf("\nGPU result = %d\n", (int)gpu_result);
	printf("CPU result = %d\n\n", (int)cpu_result);
	cout << "reduceTime = " << reduceTime << endl;

	// cleanup
	sdkDeleteTimer(&timer);
	free(h_idata);
	free(h_odata);

	checkCudaErrors(hipFree(d_idata));
	checkCudaErrors(hipFree(d_odata));

	if (gpu_result == cpu_result)
		cout << "pass!" << endl;

	return 0;
}